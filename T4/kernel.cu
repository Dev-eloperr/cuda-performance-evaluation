
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <string>

#include "VectorAdditionBenchmarker.h"
#include "FastFourierTransformBenchmarker.h"
#include "LinearSearch.h"
#include "GrayscaleBenchmarker.h"

namespace {
	template <typename T>
	void runBenchmark(std::string benchmarkName) {
		std::vector <std::pair<size_t, size_t>> benchmarks;
		benchmarks.push_back(std::make_pair(10000, 500));
		benchmarks.push_back(std::make_pair(10000, 5000));
		benchmarks.push_back(std::make_pair(10000, 50000));
		benchmarks.push_back(std::make_pair(10000, 100000));
		for (auto& i : benchmarks) {
			std::cout << "Running " << benchmarkName << " with array size " << i.second << std::endl;
			T t(i.first, i.second);
			t.benchmark();
		}
	}
}

int main() {
	runBenchmark<VectorAdditionBenchmarker>("Vector addition");
	runBenchmark<FastFourierTransformBenchmarker>("Fast fourier transform");
	runBenchmark<LinearSearch>("Linear Search");
	std::cout  << "Running Conversion from RGB to Grayscale" << std::endl;
	GrayscaleBenchmarker grayscale(50, 1);
	grayscale.benchmark();
}

