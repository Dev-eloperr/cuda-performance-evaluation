#include "hip/hip_runtime.h"
#include ""

#include "FastFourierTransformBenchmarker.h"
#include <hipfft/hipfft.h>
#include <fftw3.h>

FastFourierTransformBenchmarker::FastFourierTransformBenchmarker(size_t size, size_t arraySize)
	: arraySize(arraySize), Benchmarker(size)
{
}


std::chrono::high_resolution_clock::duration FastFourierTransformBenchmarker::runGpu()
{
	hipfftHandle plan;
	hipfftComplex *data;
	const auto NX = arraySize;
	hipMalloc(&data, sizeof(hipfftComplex) * NX );

	hipfftPlan1d(&plan, NX, HIPFFT_C2C, 10);
	auto startTime = std::chrono::high_resolution_clock::now();
	hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	auto endTime = std::chrono::high_resolution_clock::now();
	hipfftDestroy(plan);
	hipFree(data);
	return endTime - startTime;
}

std::chrono::high_resolution_clock::duration FastFourierTransformBenchmarker::runCpu()
{
	fftw_complex *in, *out;
	fftw_plan p;
	const auto N = arraySize;
	in = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * N);
	out = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * N);
	p = fftw_plan_dft_1d(N, in, out, FFTW_FORWARD, FFTW_ESTIMATE);
	auto startTime = std::chrono::high_resolution_clock::now();
	fftw_execute(p); /* repeat as needed */
	auto endTime = std::chrono::high_resolution_clock::now();
	fftw_destroy_plan(p);
	fftw_free(in); fftw_free(out);
	return endTime - startTime;
}


FastFourierTransformBenchmarker::~FastFourierTransformBenchmarker()
{
}
