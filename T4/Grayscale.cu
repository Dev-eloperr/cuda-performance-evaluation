#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv/cv.h>
#include <opencv/highgui.h>
#include <iostream>
#include <chrono>

#include "GrayscaleBenchmarker.h"

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }


    err = hipDeviceSynchronize();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

__global__ void rgb_2_grey(uchar* const greyImage, const uchar4* const rgbImage, int rows, int columns)
{
    int rgb_x = blockIdx.x * blockDim.x + threadIdx.x; //x coordinate of pixel
    int rgb_y = blockIdx.y * blockDim.y + threadIdx.y; //y coordinate of pixel

    if ((rgb_x >= columns) || (rgb_y >= rows)) {
        return;
    }

    int rgb_ab = rgb_y*columns + rgb_x; //absolute pixel position
    uchar4 rgb_Img = rgbImage[rgb_ab];
    greyImage[rgb_ab] = uchar((float(rgb_Img.x))*0.299f + (float(rgb_Img.y))*0.587f + (float(rgb_Img.z))*0.114f);
}

using namespace cv;
using namespace std;

void Proc_Img_Gpu(uchar4** h_RGBImage, uchar** h_greyImage, uchar4 **d_RGBImage, uchar** d_greyImage);
std::chrono::high_resolution_clock::duration RGB_2_Greyscale(uchar* const d_greyImage, uchar4* const d_RGBImage, size_t num_Rows, size_t num_Cols);
void Save_Img();

Mat img_RGB;
Mat img_Grey;
uchar4 *d_rgbImg;
uchar *d_greyImg; 

std::chrono::high_resolution_clock::duration GrayscaleBenchmarker::runGpu()
{
        uchar4* h_rgbImg;
        uchar* h_greyImg;

        Proc_Img_Gpu(&h_rgbImg, &h_greyImg, &d_rgbImg, &d_greyImg);
        auto time = RGB_2_Greyscale(d_greyImg, d_rgbImg, img_RGB.rows, img_RGB.cols);
        Save_Img();
	return time;
}

std::chrono::high_resolution_clock::duration GrayscaleBenchmarker::runCpu()
{
	Mat image = imread("./test.png");
	Mat gray;
	auto start = std::chrono::high_resolution_clock::now();
	cvtColor(image,gray,CV_BGR2GRAY);
	auto end = std::chrono::high_resolution_clock::now();
	return end - start;
}

GrayscaleBenchmarker::GrayscaleBenchmarker(size_t size, size_t arraySize) : Benchmarker(size) {
}

void Proc_Img_Gpu(uchar4** h_RGBImage, uchar** h_greyImage, uchar4 **d_RGBImage, uchar** d_greyImage){
    hipFree(0);
    CudaCheckError();

    //loads image into a matrix object along with the colors in BGR format (must convert to rgb).
    Mat img = imread("./test.png", CV_LOAD_IMAGE_COLOR);
    if (img.empty()){
        cerr << "couldnt open file" << endl;
        exit(1);
    }

    //converts color type from BGR to RGB
    cvtColor(img, img_RGB, CV_BGR2RGBA);

    //allocate memory for new greyscale image. 
    //img.rows returns the range of pixels in y, img.cols returns range of pixels in x
    //CV_8UC1 means 8 bit unsigned(non-negative) single channel of color, aka greyscale.
    //all three of the parameters allow the create function in the Mat class to determine how much memory to allocate
    img_Grey.create(img.rows, img.cols, CV_8UC1);

    //creates rgb and greyscale image arrays
    *h_RGBImage = (uchar4*)img_RGB.ptr<uchar>(0); //.ptr is a method in the mat class that returns a pointer to the first element of the matrix.
    *h_greyImage = (uchar*)img_Grey.ptr<uchar>(0);        //this is just like a regular array/pointer mem address to first element of the array. This is templated
                                                          //in this case the compiler runs the function for returning pointer of type unsigned char. for rgb image it is
                                                          //cast to uchar4 struct to hold r,g, and b values.

    const size_t num_pix = (img_RGB.rows) * (img_RGB.cols); //amount of pixels 

    //allocate memory on gpu
    hipMalloc(d_RGBImage, sizeof(uchar4) * num_pix); //bites of 1 uchar4 times # of pixels gives number of bites necessary for array
    CudaCheckError();
    hipMalloc(d_greyImage, sizeof(uchar) * num_pix);//bites of uchar times # pixels gives number of bites necessary for array
    CudaCheckError();
    hipMemset(*d_greyImage, 0, sizeof(uchar) * num_pix);
    CudaCheckError();


    //copy array into allocated space
    hipMemcpy(*d_RGBImage, *h_RGBImage, sizeof(uchar4)*num_pix, hipMemcpyHostToDevice);
    CudaCheckError();


    d_rgbImg = *d_RGBImage;
    d_greyImg = *d_greyImage; 
}


std::chrono::high_resolution_clock::duration RGB_2_Greyscale(uchar* const d_greyImage, uchar4* const d_RGBImage, size_t num_Rows, size_t num_Cols){
    const int BS = 16;
    const dim3 blockSize(BS, BS);
    const dim3 gridSize((num_Cols / BS) + 1, (num_Rows / BS) + 1); 

    hipDeviceSynchronize();
    auto start = chrono::high_resolution_clock::now();
    rgb_2_grey <<<gridSize, blockSize>>>(d_greyImage, d_RGBImage, num_Rows, num_Cols);
    hipDeviceSynchronize();
    auto end = chrono::high_resolution_clock::now();

    hipDeviceSynchronize(); CudaCheckError();
    return end - start;
}



void Save_Img(){

    const size_t num_pix = (img_RGB.rows) * (img_RGB.cols);
    hipMemcpy(img_Grey.ptr<uchar>(0), d_greyImg, sizeof(uchar)*num_pix, hipMemcpyDeviceToHost);
    CudaCheckError();

    hipFree(d_rgbImg);
    hipFree(d_greyImg);

}

